#include "hip/hip_runtime.h"
#include "Network.h"

#include <fstream>
#include <iostream>
#include <filesystem/path.h>

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <json/json.hpp>

using namespace tcnn;
using precision_t = network_precision_t;//冒红光是因为这个写在条件编译里的

namespace
{

struct NetworkComponents {
    std::shared_ptr<Loss<precision_t>> loss = nullptr;
    std::shared_ptr<Optimizer<precision_t>> optimizer = nullptr;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = nullptr;
    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer = nullptr;
};

struct IOData {
    GPUMatrix<float>* input_mat = nullptr;
    GPUMatrix<float>* output_mat = nullptr;

    GPUMatrixDynamic<float>* training_input_mat = nullptr;
    GPUMatrix<float>* training_output_mat = nullptr;
};

hipStream_t inference_stream = nullptr;
hipStream_t training_stream = nullptr;

NetworkComponents* mNetworkComponents = nullptr;

IOData* mIOData = nullptr;

}

template <typename T, uint32_t stride>
__global__ void formatInput(uint32_t n_elements, Falcor::RadiosityQuery* queries, T* input)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    Falcor::RadiosityQuery query = queries[i];

    input[i * stride + 0] = query.posW.x;           input[i * stride + 1] = query.posW.y;           input[i * stride + 2] = query.posW.z;
    input[i * stride + 3] = query.normalW.x;        input[i * stride + 4] = query.normalW.y;        input[i * stride + 5] = query.normalW.z;
    input[i * stride + 6] = query.wiW.x;            input[i * stride + 7] = query.wiW.y;            input[i * stride + 8] = query.wiW.z;
    input[i * stride + 9] = query.diff.x;           input[i * stride + 10] = query.diff.y;          input[i * stride + 11] = query.diff.z;
}


template <typename T, uint32_t stride>
__global__ void mapToOutSurf(uint32_t n_elements, uint32_t width, T* output, hipSurfaceObject_t outSurf)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    uint32_t x = i % width;
    uint32_t y = i / width;

    float4 color = { 0, 0, 0, 1 };

    color.x = output[i * stride + 0];
    color.y = output[i * stride + 1];
    color.z = output[i * stride + 2];

    surf2Dwrite(color, outSurf, x * sizeof(float4), y);
}


RadiosityNetwork::RadiosityNetwork(const uint32_t width, const uint32_t height)
{
    CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
    CUDA_CHECK_THROW(hipStreamCreate(&training_stream));

    mNetworkComponents = new NetworkComponents();
    mIOData = new IOData();

    filesystem::path c_path(NetConfig::netConfigPath);
    if (!c_path.exists()) {
        std::cout << "Cannot find the network config!" << std::endl;
        return;
    } else {
        std::cout << "Successfully find the network config!" << std::endl;
    }

    std::ifstream f(c_path.str());
    json config = json::parse(f, nullptr, true, true);

    json encoding_opts = config.value("encoding", json::object());
	json loss_opts = config.value("loss", json::object());
	json optimizer_opts = config.value("optimizer", json::object());
	json network_opts = config.value("network", json::object());

    mNetworkComponents->loss = std::shared_ptr<Loss<precision_t>>(create_loss<precision_t>(loss_opts));
    mNetworkComponents->optimizer = std::shared_ptr<Optimizer<precision_t>>(create_optimizer<precision_t>(optimizer_opts));
    mNetworkComponents->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(NetConfig::n_input_dims, NetConfig::n_output_dims, encoding_opts, network_opts);
    mNetworkComponents->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(mNetworkComponents->network, mNetworkComponents->optimizer, mNetworkComponents->loss);

    filesystem::path w_path(NetConfig::weightsPath);
    if (!w_path.exists()) {
        std::cout << "Cannot find the weights!" << std::endl;
        return;
    } else {
        std::cout << "Successfully find the weights!" << std::endl;
    }
    std::ifstream wf(w_path.str());
    json loaded_weights = json::parse(wf, nullptr, true, true);

    //下面注释掉了还能跑，我感觉是要注释掉的，应该换成训练好的weight但是好像也没有用这个就是了。
    //但是注释掉不注释掉渲染出的图像是不一样的，所以反正之前是用了权重数据的
    // 按理说是没用这个，如果发现一定要走这个的话可以实时读写
    //mNetworkComponents->trainer->deserialize(loaded_weights);

    mIOData->input_mat = new GPUMatrix<float>(NetConfig::n_input_dims, width * height);
    mIOData->output_mat = new GPUMatrix<float>(NetConfig::n_output_dims, width * height);
    mIOData->training_input_mat = new GPUMatrix<float>(NetConfig::n_input_dims, width * height);
    mIOData->training_output_mat = new GPUMatrix<float>(NetConfig::n_output_dims, width * height);


    frame_width = width;
    frame_height = height;
}


RadiosityNetwork::~RadiosityNetwork()
{
    delete mNetworkComponents;
    delete mIOData;
}


void RadiosityNetwork::forward(Falcor::RadiosityQuery* queries, hipSurfaceObject_t output)
{
    uint32_t n_elements = frame_width * frame_height;

    //将查询数据（queries）转换为神经网络的输入张量
    linear_kernel(formatInput<float, NetConfig::n_input_dims>, 0, inference_stream, n_elements, queries, mIOData->input_mat->data());

    mNetworkComponents->network->inference(inference_stream, *mIOData->input_mat, *mIOData->output_mat);

    //将网络推断的结果映射或渲染到输出表面
    linear_kernel(mapToOutSurf<float, NetConfig::n_output_dims>, 0, inference_stream, n_elements, frame_width, mIOData->output_mat->data(), output);
} 

//这里pada一些权重文件会改吗，我吐了
//这个数据到底是从trainingstream里读还是queries里读的啊
//这个linear_kernel应该是用来转成可以放到网络里的形式吧
void RadiosityNetwork::train(Falcor::RadiosityQuery* queries, hipSurfaceObject_t targets, float& loss)
{
    uint32_t n_elements = frame_width * frame_height;
    mNetworkComponents->optimizer->set_learning_rate(learning_rate);

    /// self query,大胆猜测这个和上面的forward前两句一个道理
    //linear_kernel(
    //    formatInput<float, NetConfig::n_input_dims>,
    //    0,
    //    training_stream,
    //    self_query_batch_size,
    //    0,
    //    self_queries,
    //    mMemory->training_self_query->data()
    //);
    //mNetwork->network->inference(training_stream, *mMemory->training_self_query, *mMemory->training_self_pred);
    //上面的mMemory->training_self_query就是*mIOData->input_mat，*mMemory->training_self_pre就是*mIOData->output_mat

    linear_kernel(formatInput<float, NetConfig::n_input_dims>, 0, training_stream, n_elements, queries, mIOData->input_mat->data());
    mNetworkComponents->network->inference(training_stream, *mIOData->input_mat, *mIOData->output_mat);

    mNetworkComponents->trainer->training_step(training_stream, *mIOData->input_mat, *mIOData->output_mat);
    //下面的运行了，他只是不更新网络权重罢了。上面的最后一个输入参数改成input他就会闪退，所以我感觉是执行了的
    //std::cout << "Hello World!" << std::endl;
    // 确保所有CUDA操作都已完成
    CUDA_CHECK_THROW(hipStreamSynchronize(training_stream));
}
