#include "hip/hip_runtime.h"
#include "Network.h"

#include <fstream>
#include <iostream>
#include <filesystem/path.h>

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <json/json.hpp>

using namespace tcnn;
using precision_t = network_precision_t;

namespace
{

struct NetworkComponents
{
    std::shared_ptr<Loss<precision_t>> loss = nullptr;
    std::shared_ptr<Optimizer<precision_t>> optimizer = nullptr;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = nullptr;
    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer = nullptr;
};

struct IOData
{
    GPUMatrix<float>* render_input_mat = nullptr;
    GPUMatrix<float>* render_output_mat = nullptr;

    GPUMatrix<float>* training_input_mat = nullptr;
    GPUMatrix<float>* training_output_mat = nullptr;
};

hipStream_t inference_stream = nullptr;
hipStream_t training_stream = nullptr;


NetworkComponents* mNetworkComponents = nullptr;

IOData* mIOData = nullptr;
json loaded_weights;

} // namespace


uint32_t showMsg_counter(uint32_t* dataOnDevice)
{
    uint32_t* dataOnHost = new uint32_t[1];
    hipMemcpy(dataOnHost, dataOnDevice, sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("%u\n", dataOnHost[0]);
    uint32_t res = dataOnHost[0];
    delete[] dataOnHost;
    return res;
}


template<typename T, uint32_t input_stride, uint32_t output_stride>
__global__ void formatInputTarget(uint32_t n_elements, Falcor::RadianceQuery* queries, Falcor::RadianceTarget* targets,
    T* input, T* output, uint32_t* trainCount)
{
    n_elements = *trainCount;//woc居然只有这样可以读到，根本不能在外面读到一点儿
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements)
        return;
    

    Falcor::RadianceQuery query = queries[i];
    Falcor::RadianceTarget target = targets[i];

    input[i * input_stride + 0] = query.pos.x, input[i * input_stride + 1] = query.pos.y, input[i * input_stride + 2] = query.pos.z;
    input[i * input_stride + 3] = query.dir.x, input[i * input_stride + 4] = query.dir.y;

    input[i * input_stride + 5] = query.roughness;
    input[i * input_stride + 6] = query.normal.x, input[i * input_stride + 7] = query.normal.y;
    input[i * input_stride + 8] = query.diffuse.x, input[i * input_stride + 9] = query.diffuse.y, input[i * input_stride + 10] = query.diffuse.z;
    input[i * input_stride + 11] = query.specular.x, input[i * input_stride + 12] = query.specular.y,
    input[i * input_stride + 13] = query.specular.z;

    output[i * output_stride + 0] = target.radiance.x, output[i * output_stride + 1] = target.radiance.y, output[i * output_stride + 2] = target.radiance.z;

   
}

template<typename T, uint32_t input_stride>
__global__ void formatRenderInput(uint32_t n_elements, Falcor::RadianceQuery* queries, T* input)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements)
        return;

    Falcor::RadianceQuery query = queries[i];

    input[i * input_stride + 0] = query.pos.x, input[i * input_stride + 1] = query.pos.y, input[i * input_stride + 2] = query.pos.z;
    input[i * input_stride + 3] = query.dir.x, input[i * input_stride + 4] = query.dir.y;

    input[i * input_stride + 5] = query.roughness;
    input[i * input_stride + 6] = query.normal.x, input[i * input_stride + 7] = query.normal.y;
    input[i * input_stride + 8] = query.diffuse.x, input[i * input_stride + 9] = query.diffuse.y, input[i * input_stride + 10] = query.diffuse.z;
    input[i * input_stride + 11] = query.specular.x, input[i * input_stride + 12] = query.specular.y, input[i * input_stride + 13] = query.specular.z;
}

template<typename T, uint32_t stride>
__global__ void mapToOutSurf(uint32_t n_elements, uint32_t width, T* output, hipSurfaceObject_t outSurf)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements)
        return;

    uint32_t x = i % width;
    uint32_t y = i / width;

    float4 color = {0, 0, 0, 1};

    color.x = output[i * stride + 0];
    color.y = output[i * stride + 1];
    color.z = output[i * stride + 2];

    surf2Dwrite(color, outSurf, x * sizeof(float4), y);
}


NRCNetwork :: NRCNetwork(const uint32_t width, const uint32_t height)
{
    std::cout << "Hello World!" << width << height << std::endl;

    CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
    CUDA_CHECK_THROW(hipStreamCreate(&training_stream));

    mNetworkComponents = new NetworkComponents();
    mIOData = new IOData();

    filesystem::path c_path(NetConfig::netConfigPath);
    if (!c_path.exists())
    {
        std::cout << "Cannot find the network config!" << std::endl;
        return;
    }
    else
    {
        std::cout << "Successfully find the network config!" << std::endl;
    }

    std::ifstream f(c_path.str());
    json config = json::parse(f, nullptr, true, true);

    json encoding_opts = config.value("encoding", json::object());
    json loss_opts = config.value("loss", json::object());
    json optimizer_opts = config.value("optimizer", json::object());
    json network_opts = config.value("network", json::object());

    mNetworkComponents->loss = std::shared_ptr<Loss<precision_t>>(create_loss<precision_t>(loss_opts));
    mNetworkComponents->optimizer = std::shared_ptr<Optimizer<precision_t>>(create_optimizer<precision_t>(optimizer_opts));
    mNetworkComponents->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(
        NetConfig::n_input_dims, NetConfig::n_output_dims, encoding_opts, network_opts
    );
    mNetworkComponents->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(
        mNetworkComponents->network, mNetworkComponents->optimizer, mNetworkComponents->loss
    );

    filesystem::path w_path(NetConfig::weightsPath);
    if (!w_path.exists())
    {
        std::cout << "Cannot find the weights!" << std::endl;
        return;
    }
    else
    {
        std::cout << "Successfully find the weights!" << std::endl;
    }
    std::ifstream wf(w_path.str());
    json loaded_weights = json::parse(wf, nullptr, true, true);


    mIOData->render_input_mat = new GPUMatrix<float>(NetConfig::n_input_dims, width * height);
    mIOData->render_output_mat = new GPUMatrix<float>(NetConfig::n_output_dims, width * height);
    mIOData->training_input_mat = new GPUMatrix<float>(NetConfig::n_input_dims, width * height);
    mIOData->training_output_mat = new GPUMatrix<float>(NetConfig::n_output_dims, width * height);

    frame_width = width;
    frame_height = height;
}

NRCNetwork::~NRCNetwork()
{
    delete mNetworkComponents;
    delete mIOData;
}

void NRCNetwork::Test()
{
    //std::cout << "Hello World!" << std::endl;
}


void NRCNetwork ::train(Falcor::RadianceQuery* queries, Falcor::RadianceTarget* targets, float& loss, uint32_t* trainCounts)
{
    //std::cout << "Hello World!" << std::endl;
    //uint32_t n_elements = trainCounts[0].trainCounter;//targets能读到这里怎么会读不到呢
    //uint32_t n_elements = *trainCounts;
    uint32_t n_elements = 3 * frame_width * frame_height;
    mNetworkComponents->optimizer->set_learning_rate(learning_rate);

    linear_kernel(formatInputTarget<float, NetConfig::n_input_dims, NetConfig::n_output_dims>, 0, training_stream, n_elements, queries, targets,
        mIOData->training_input_mat->data(), mIOData->training_output_mat->data(), trainCounts
    );

    //std::cout << "input[i * stride + 0]" << mIOData->training_input_mat->data() << std::endl;
    //std::cout << "output[i * stride + 0]" << mIOData->training_output_mat->data() << std::endl;
    uint32_t temp = showMsg_counter(trainCounts);//一直是0，xsl
    //我真的不理解为什么w加了这个，应该是如果网络里放过了就不重复喂了？
    //mNetworkComponents->network->inference(training_stream, *mIOData->training_input_mat, *mIOData->training_output_mat);
    auto ctx = mNetworkComponents->trainer->training_step(training_stream, *mIOData->training_input_mat, *mIOData->training_output_mat);
    float tmp_loss = 0;
    tmp_loss = mNetworkComponents->trainer->loss(training_stream, *ctx);
    //std::cout << tmp_loss << std::endl;
    //json loaded_weights;
    //loaded_weights = mNetworkComponents->trainer->serialize(false);
    //std::cout << loaded_weights.dump(4) << std::endl;
    //if (loaded_weights) std::cout << "Hello World!" << std::endl;
    //else std::cout << "MD World!" << std::endl;
    //std::string network_config_save_path = "network_weights.json";
    //std::ofstream of(network_config_save_path);
    //of << loaded_weights.dump(4);
    //of.close();
    CUDA_CHECK_THROW(hipStreamSynchronize(training_stream));
}



void NRCNetwork ::forward(Falcor::RadianceQuery* queries, hipSurfaceObject_t output)
{
    //json loaded_weights;
    //loaded_weights = mNetworkComponents->trainer->serialize(false);
    //std::cout << loaded_weights.dump(4) << std::endl;

    uint32_t n_elements = frame_width * frame_height;
    //mNetworkComponents->trainer->deserialize(loaded_weights);
    linear_kernel(formatRenderInput<float, NetConfig::n_input_dims>, 0, inference_stream, n_elements, queries, mIOData->render_input_mat->data());
    mNetworkComponents->network->inference(inference_stream, *mIOData->render_input_mat, *mIOData->render_output_mat);
    linear_kernel(
        mapToOutSurf<float, NetConfig::n_output_dims>, 0, inference_stream, n_elements, frame_width,
        mIOData->render_output_mat->data(), output
    );
    CUDA_CHECK_THROW(hipStreamSynchronize(inference_stream));
}

